#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__

// 并行实现矩阵组装
// 通过索引线程号决定如何计算
__device__
void MatrixAssemble(double[] matrixA, double[] vectorB, int N) {
    hipMalloc(&d_matrixA, N*N*sizeof(double));
    hipMalloc(&d_vectorB, N*sizeof(double));
    const int threadIndexX = threadIdx.x + blockIdx.x * blockDim.x;
    const int threadIndexY = threadIdx.y + blockIdx.y * blockDim.y;
    if(blablabla) {
        d_matrixA[] = GaussIntegral(&LeftFunction, &der_1_Basis1, &der_1_Basis2, GaussInterval);
        d_vectorB[] = GaussIntegral(&RightFunction, &Basis1, &Basis2, GaussInterval);
    }
}