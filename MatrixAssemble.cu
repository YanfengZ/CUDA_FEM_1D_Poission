#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include 'GaussIntegral'
#include 'Leftfunction'
#include 'RightFunction'

__global__

// 并行实现矩阵组装
// 通过索引线程号决定如何计算
__device__
void MatrixAssemble(double[] matrixA, double[] vectorB, int N) {
    hipMalloc(&d_matrixA, N*N*sizeof(double));
    hipMalloc(&d_vectorB, N*sizeof(double));
    const int threadIndexX = threadIdx.x + blockIdx.x * blockDim.x;
    const int threadIndexY = threadIdx.y + blockIdx.y * blockDim.y;
    if(blablabla) {
        d_matrixA[] = GaussIntegral();
        d_vectorB[] = GaussIntegral();
    }
}